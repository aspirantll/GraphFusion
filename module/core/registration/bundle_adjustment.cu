#include "hip/hip_runtime.h"
//
// Created by liulei on 2020/6/12.
//

#include "bundle_adjustment.cuh"

namespace rtf {
    constexpr Scalar kHuberWeight = 1.2;
    __device__ Scalar computeHuberWeight(Scalar residual_x, Scalar residual_y, Scalar huber_parameter) {
        Scalar squared_residual = residual_x * residual_x + residual_y * residual_y;
        return (squared_residual < huber_parameter * huber_parameter) ? 1 : (huber_parameter / sqrtf(squared_residual));
    }

    __device__ Scalar ComputeHuberCost(Scalar residual_x, Scalar residual_y, Scalar huber_parameter) {
        Scalar squared_residual = residual_x * residual_x + residual_y * residual_y;
        if (squared_residual < huber_parameter * huber_parameter) {
            return 0.5 * squared_residual;
        } else {
            return huber_parameter * (sqrtf(squared_residual) - 0.5 * huber_parameter);
        }
    }

    __device__ void projectJacobi(float3x3 k, Scalar* point, Scalar* proJacobi) {
        Scalar x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        Scalar y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        Scalar z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        proJacobi[0] = z*k(0,0)-x*k(2,0)/(z*z);
        proJacobi[1] = z*k(0,1)-x*k(2,1)/(z*z);
        proJacobi[2] = z*k(0,2)-x*k(2,2)/(z*z);
        proJacobi[3] = z*k(1,0)-y*k(2,0)/(z*z);
        proJacobi[4] = z*k(1,1)-y*k(2,1)/(z*z);
        proJacobi[5] = z*k(1,2)-y*k(2,2)/(z*z);
    }

    __device__ void transformPointOnlyRotation(float4x4 T, Scalar* point, Scalar *tPoint) {
        Scalar x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2];
        Scalar y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2];
        Scalar z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2];

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }


    __device__ void transformPoint(float4x4 T, Scalar* point, Scalar *tPoint) {
        Scalar x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2] + T(0,3);
        Scalar y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2] + T(1,3);
        Scalar z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2] + T(2,3);

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }

    __device__ void projectPoint(float3x3 k, Scalar* point, Scalar* pixel) {
        Scalar x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        Scalar y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        Scalar z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        pixel[0] = x/z;
        pixel[1] = y/z;
    }

    __device__ void hatMatrix(Scalar* point, Scalar* hat) {
        hat[0] = 0;
        hat[1] = -point[2];
        hat[2] = point[1];
        hat[3] = point[2];
        hat[4] = 0;
        hat[5] = -point[0];
        hat[6] = -point[1];
        hat[7] = point[0];
        hat[8] = 0;
    }

    // x = u/fx*d-cx/fx*d, y=v/fy*d-cy/fy*d, z=d
    __device__ void unproject(float3x3 K, Scalar* pixel, Scalar *dst) {
        Scalar fx=K(0, 0), fy=K(1, 1), cx=K(0, 2), cy=K(1, 2);
        dst[0] = pixel[2]*(pixel[0]-cx)/fx;
        dst[1] = pixel[2]*(pixel[1]-cy)/fy;
        dst[2] = pixel[2];
    }

    __device__ void computeJacobi(Scalar* proJacobi, Scalar* hat, Scalar* jacobi) {
        // for R
        jacobi[0] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[1] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[2] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[6] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[7] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[8] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);

        // for t
        jacobi[3] = proJacobi[0];
        jacobi[4] = proJacobi[1];
        jacobi[5] = proJacobi[2];

        jacobi[9] = proJacobi[3];
        jacobi[10] = proJacobi[4];
        jacobi[11] = proJacobi[5];
    }


    __device__ void computeDeltaLie(Scalar *jacobi, Scalar *residual, Scalar* delta) {
        delta[0] = fabs(residual[0]/jacobi[0]+residual[1]/jacobi[6]);
        delta[1] = fabs(residual[0]/jacobi[1]+residual[1]/jacobi[7]);
        delta[2] = fabs(residual[0]/jacobi[2]+residual[1]/jacobi[8]);
        delta[3] = fabs(residual[0]/jacobi[3]+residual[1]*jacobi[9]); // jacobi is 0
        delta[4] = fabs(residual[0]*jacobi[4]+residual[1]/jacobi[10]);
        delta[5] = fabs(residual[0]/jacobi[5]+residual[1]/jacobi[11]);

    }



    __global__ void computeCostAndJacobi(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator, CUDAPtrs hSummator, CUDAPtrs mSummator, CUDAPtrs bSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;
        if(mask[index]) {
            Scalar point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)}
            , rePixel[2], transPoint[3], rotatePoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
            // copy point and pixel
            transformPoint(T, point, transPoint);

            // compute jacobi
            projectJacobi(K, transPoint, proJacobi);
            transformPointOnlyRotation(T, point, rotatePoint);
            hatMatrix(rotatePoint, hatMat);
            computeJacobi(proJacobi, hatMat, jacobi);

            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            Scalar weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
            Scalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
            // compute H,M,b
            Scalar * H = hSummator.data+index*36;
            Scalar * M = mSummator.data+index*6;
            Scalar * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
                }
                M[i] = weight*H[i*6+i];
                b[i] = -weight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
            }
        }else {
            // compute H,M,b
            costSummator.data[index]=0;
            Scalar * H = hSummator.data+index*36;
            Scalar * M = mSummator.data+index*6;
            Scalar * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = 0;
                }
                M[i] = 0;
                b[i] = 0;
            }
        }

    }

    __global__ void computeCost(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;

        if(mask[index]) {
            Scalar point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)};

            Scalar rePixel[2], transPoint[3], residual[2];
            transformPoint(T, point, transPoint);
            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            Scalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
        }else {
            costSummator.data[index] = 0;
        }
    }


    void computeBACostAndJacobi(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator, Summator& hSummator, Summator& mSummator, Summator& bSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCostAndJacobi<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat, *hSummator.dataMat, *mSummator.dataMat, *bSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    void computeBACost(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCost<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    __global__ void computerInliersKernel(CUDAPtrs cost, CUDAPtrc inliers, Scalar th) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=cost.getRows()) return;
        inliers.setIndex(index, cost[index]>0&&cost[index]<th);
    }

    void computerInliers(Summator& costSummator, CUDAMatrixc& inliers, Scalar th) {
        // invoke kernel
        CUDA_LINE_BLOCK(costSummator.length);

        computerInliersKernel<<<grid, block, 0, stream>>>(*costSummator.dataMat, inliers, th);

        CUDA_CHECKED_NO_ERROR();
    }

    __device__ void composeJacobi(Scalar* proJacobi, Scalar* hat, Scalar* jacobi) {
        // for t
        jacobi[0] = proJacobi[0];
        jacobi[1] = proJacobi[1];
        jacobi[2] = proJacobi[2];

        jacobi[6] = proJacobi[3];
        jacobi[7] = proJacobi[4];
        jacobi[8] = proJacobi[5];

        // for R
        jacobi[3] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[4] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[5] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[9] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[10] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[11] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);
    }

    __device__ void computeHMb(Scalar* H, Scalar* M, Scalar* b, Scalar weight, Scalar* jacobi, Scalar* residual) {
        for(int i=0; i<6; i++) {
            for(int j=0; j<6; j++) {
                H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
            }
            M[i] = weight*H[i*6+i];
            b[i] = -weight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
        }
    }


    __global__ void computeMVCostAndJacobiForEdge(CUDAEdge edge, CUDAPtrs H, CUDAPtrs M, CUDAPtrs b, Scalar* cost) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        const float3x3 intrinsicX = edge.intrinsicX;
        const float3x3 intrinsicY = edge.intrinsicY;
        const float4x4 transform = edge.transform;
        const int x = edge.indexX;
        const int y = edge.indexY;
        const int z = edge.indexZ;
        const int n = H.rows;

        if(index>=kx.getRows()) return;

        Scalar point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};
        Scalar rePixel[2], transPoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        // compute jacobi
        projectJacobi(intrinsicX, transPoint, proJacobi);
        hatMatrix(transPoint, hatMat);
        composeJacobi(proJacobi, hatMat, jacobi);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        Scalar weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
        Scalar huberCost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

        // compute H,M,b
        Scalar tH[36], tM[6], tb[6];
        computeHMb(tH, tM, tb, weight, jacobi, residual);

        for(int i=0; i<6; i++) {
            for(int j=0; j<6; j++) {
                Scalar value = tH[6*j+i];
                atomicAdd(&H.data[(6*x+j)*n+i+6*x], value);
                atomicAdd(&H.data[(6*x+j)*n+i+6*y], -value);
                atomicAdd(&H.data[(6*x+j)*n+i+6*z], -value);

                atomicAdd(&H.data[(6*y+j)*n+i+6*x], -value);
                atomicAdd(&H.data[(6*y+j)*n+i+6*y], value);
                atomicAdd(&H.data[(6*y+j)*n+i+6*z], value);

                atomicAdd(&H.data[(6*z+j)*n+i+6*x], -value);
                atomicAdd(&H.data[(6*z+j)*n+i+6*y], value);
                atomicAdd(&H.data[(6*z+j)*n+i+6*z], value);
            }
            atomicAdd(&M.data[6*x+i], -tM[i]);
            atomicAdd(&M.data[6*y+i], tM[i]);
            atomicAdd(&M.data[6*z+i], tM[i]);

            atomicAdd(&b.data[6*x+i], -tb[i]);
            atomicAdd(&b.data[6*y+i], tb[i]);
            atomicAdd(&b.data[6*z+i], tb[i]);
        }

        atomicAdd(cost, huberCost);
    }

    __global__ void computeMVCostForEdge(CUDAEdge edge, Scalar* cost) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        float3x3 intrinsicX = edge.intrinsicX;
        float3x3 intrinsicY = edge.intrinsicY;
        float4x4 transform = edge.transform;

        if(index>=kx.getRows()) return;

        Scalar point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};

        Scalar rePixel[2], transPoint[3], residual[2];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        atomicAdd(cost, ComputeHuberCost(residual[0], residual[1], kHuberWeight));
    }


    void computeMVBACostAndJacobi(CUDAEdgeVector &edges, LMSumMats& sumMats) {
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].kx.rows);

            computeMVCostAndJacobiForEdge<<<grid, block, 0, stream>>>(edges[index], *sumMats.cH, *sumMats.cM, *sumMats.cb, sumMats.cCost);

            CUDA_CHECKED_NO_ERROR();
        }
    }

    void computeMVBACost(CUDAEdgeVector &edges, Scalar& cost) {
        cost = 0;
        Scalar *cCost;
        CUDA_CHECKED_CALL(hipMalloc(&cCost, sizeof(Scalar)));
        CUDA_CHECKED_CALL(hipMemcpy(cCost, &cost, sizeof(Scalar), hipMemcpyHostToDevice));
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].kx.rows);

            computeMVCostForEdge<<<grid, block, 0, stream>>>(edges[index], cCost);

            CUDA_CHECKED_NO_ERROR();
        }
        CUDA_CHECKED_CALL(hipMemcpy(&cost, cCost, sizeof(Scalar), hipMemcpyDeviceToHost));
        CUDA_CHECKED_CALL(hipFree(cCost));
    }

}
