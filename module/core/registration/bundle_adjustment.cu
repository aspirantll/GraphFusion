#include "hip/hip_runtime.h"
//
// Created by liulei on 2020/6/12.
//

#include "bundle_adjustment.cuh"

namespace rtf {
    constexpr float kHuberWeight = 1.2;
    __device__ float computeHuberWeight(float residual_x, float residual_y, float huber_parameter) {
        float squared_residual = residual_x * residual_x + residual_y * residual_y;
        return (squared_residual < huber_parameter * huber_parameter) ? 1 : (huber_parameter / sqrtf(squared_residual));
    }

    __device__ float ComputeHuberCost(float residual_x, float residual_y, float huber_parameter) {
        float squared_residual = residual_x * residual_x + residual_y * residual_y;
        if (squared_residual < huber_parameter * huber_parameter) {
            return 0.5 * squared_residual;
        } else {
            return huber_parameter * (sqrtf(squared_residual) - 0.5 * huber_parameter);
        }
    }

    __device__ void projectJacobi(float3x3 k, float* point, float* proJacobi) {
        float x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        float y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        float z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        proJacobi[0] = z*k(0,0)-x*k(2,0)/(z*z);
        proJacobi[1] = z*k(0,1)-x*k(2,1)/(z*z);
        proJacobi[2] = z*k(0,2)-x*k(2,2)/(z*z);
        proJacobi[3] = z*k(1,0)-y*k(2,0)/(z*z);
        proJacobi[4] = z*k(1,1)-y*k(2,1)/(z*z);
        proJacobi[5] = z*k(1,2)-y*k(2,2)/(z*z);
    }

    __device__ void transformPointOnlyRotation(float4x4 T, float* point, float *tPoint) {
        float x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2];
        float y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2];
        float z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2];

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }


    __device__ void transformPoint(float4x4 T, float* point, float *tPoint) {
        float x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2] + T(0,3);
        float y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2] + T(1,3);
        float z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2] + T(2,3);

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }

    __device__ void projectPoint(float3x3 k, float* point, float* pixel) {
        float x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        float y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        float z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        pixel[0] = x/z;
        pixel[1] = y/z;
    }

    __device__ void hatMatrix(float* point, float* hat) {
        hat[0] = 0;
        hat[1] = -point[2];
        hat[2] = point[1];
        hat[3] = point[2];
        hat[4] = 0;
        hat[5] = -point[0];
        hat[6] = -point[1];
        hat[7] = point[0];
        hat[8] = 0;
    }

    // x = u/fx*d-cx/fx*d, y=v/fy*d-cy/fy*d, z=d
    __device__ void unproject(float3x3 K, float* pixel, float *dst) {
        float fx=K(0, 0), fy=K(1, 1), cx=K(0, 2), cy=K(1, 2);
        dst[0] = pixel[2]*(pixel[0]-cx)/fx;
        dst[1] = pixel[2]*(pixel[1]-cy)/fy;
        dst[2] = pixel[2];
    }

    __device__ void computeJacobi(float* proJacobi, float* hat, float* jacobi) {
        // for R
        jacobi[0] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[1] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[2] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[6] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[7] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[8] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);

        // for t
        jacobi[3] = proJacobi[0];
        jacobi[4] = proJacobi[1];
        jacobi[5] = proJacobi[2];

        jacobi[9] = proJacobi[3];
        jacobi[10] = proJacobi[4];
        jacobi[11] = proJacobi[5];
    }


    __device__ void computeDeltaLie(float *jacobi, float *residual, float* delta) {
        delta[0] = fabs(residual[0]/jacobi[0]+residual[1]/jacobi[6]);
        delta[1] = fabs(residual[0]/jacobi[1]+residual[1]/jacobi[7]);
        delta[2] = fabs(residual[0]/jacobi[2]+residual[1]/jacobi[8]);
        delta[3] = fabs(residual[0]/jacobi[3]+residual[1]*jacobi[9]); // jacobi is 0
        delta[4] = fabs(residual[0]*jacobi[4]+residual[1]/jacobi[10]);
        delta[5] = fabs(residual[0]/jacobi[5]+residual[1]/jacobi[11]);

    }



    __global__ void computeCostAndJacobi(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator, CUDAPtrs hSummator, CUDAPtrs mSummator, CUDAPtrs bSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;
        if(mask[index]) {
            float point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)}
            , rePixel[2], transPoint[3], rotatePoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
            // copy point and pixel
            transformPoint(T, point, transPoint);

            // compute jacobi
            projectJacobi(K, transPoint, proJacobi);
            transformPointOnlyRotation(T, point, rotatePoint);
            hatMatrix(rotatePoint, hatMat);
            computeJacobi(proJacobi, hatMat, jacobi);

            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            float weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
            float cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
            // compute H,M,b
            float * H = hSummator.data+index*36;
            float * M = mSummator.data+index*6;
            float * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
                }
                M[i] = weight*H[i*6+i];
                b[i] = -weight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
            }
        }else {
            // compute H,M,b
            costSummator.data[index]=0;
            float * H = hSummator.data+index*36;
            float * M = mSummator.data+index*6;
            float * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = 0;
                }
                M[i] = 0;
                b[i] = 0;
            }
        }

    }

    __global__ void computeCost(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;

        if(mask[index]) {
            float point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)};

            float rePixel[2], transPoint[3], residual[2];
            transformPoint(T, point, transPoint);
            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            float cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
        }else {
            costSummator.data[index] = 0;
        }
    }


    void computeBACostAndJacobi(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator, Summator& hSummator, Summator& mSummator, Summator& bSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCostAndJacobi<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat, *hSummator.dataMat, *mSummator.dataMat, *bSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    void computeBACost(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCost<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    __global__ void computerInliersKernel(CUDAPtrs cost, CUDAPtrc inliers, float th) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=cost.getRows()) return;
        inliers.setIndex(index, cost[index]>0&&cost[index]<th);
    }

    void computerInliers(Summator& costSummator, CUDAMatrixc& inliers, float th) {
        // invoke kernel
        CUDA_LINE_BLOCK(costSummator.length);

        computerInliersKernel<<<grid, block, 0, stream>>>(*costSummator.dataMat, inliers, th);

        CUDA_CHECKED_NO_ERROR();
    }

    __device__ void composeJacobi(float* proJacobi, float* hat, float* jacobi) {
        // for t
        jacobi[0] = proJacobi[0];
        jacobi[1] = proJacobi[1];
        jacobi[2] = proJacobi[2];

        jacobi[6] = proJacobi[3];
        jacobi[7] = proJacobi[4];
        jacobi[8] = proJacobi[5];

        // for R
        jacobi[3] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[4] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[5] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[9] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[10] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[11] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);
    }

    __device__ void computeHMb(CUDALMSummators summators, long index, float weight, float* jacobi, float* residual, float jacobiWeight) {
        float * H = summators.H.data+index*36;
        float * M = summators.M.data+index*6;
        float * b = summators.b.data+index*6;
        for(int i=0; i<6; i++) {
            for(int j=0; j<6; j++) {
                H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
            }
            M[i] = weight*H[i*6+i];
            b[i] = -weight*jacobiWeight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
        }
    }


    __global__ void computeMVCostAndJacobiForEdge(CUDAEdge edge, CUDALMSummators summatorsX, CUDALMSummators summatorsY, CUDALMSummators deltaSummators, CUDAPtrs costSummator) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        long sumIndexX = edge.sumIndexX+index;
        long sumIndexY = edge.sumIndexY+index;
        long costIndex = edge.costIndex+index;
        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        float3x3 intrinsicX = edge.intrinsicX;
        float3x3 intrinsicY = edge.intrinsicY;
        float4x4 transform = edge.transform;

        if(index>=kx.getRows()) return;

        float point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};
        float rePixel[2], transPoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        // compute jacobi
        projectJacobi(intrinsicX, transPoint, proJacobi);
        hatMatrix(transPoint, hatMat);
        composeJacobi(proJacobi, hatMat, jacobi);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        float weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
        float cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

        costSummator.data[costIndex]=cost;
//        printf("%f\n", cost);
        // compute H,M,b
        computeHMb(summatorsX, sumIndexX, weight, jacobi, residual, 1.0);
        computeHMb(summatorsY, sumIndexY, weight, jacobi, residual, -1.0);
        computeHMb(deltaSummators, index, weight, jacobi, residual, 1.0);
    }

    __global__ void computeMVCostForEdge(CUDAEdge edge, CUDAPtrs costSummator) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        long costIndex = edge.costIndex+index;
        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        float3x3 intrinsicX = edge.intrinsicX;
        float3x3 intrinsicY = edge.intrinsicY;
        float4x4 transform = edge.transform;

        if(index>=kx.getRows()) return;

        float point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};

        float rePixel[2], transPoint[3], residual[2];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        float cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

        costSummator.data[costIndex]=cost;
    }


    void computeMVBACostAndJacobi(CUDAEdgeVector &edges, CUDAVector<CUDALMSummators>& gtSummators, CUDAVector<CUDALMSummators>& deltaSummators, Summator& costSummator) {
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].count);

            computeMVCostAndJacobiForEdge<<<grid, block, 0, stream>>>(edges[index], gtSummators[edges[index].indexX], gtSummators[edges[index].indexY], deltaSummators[index], *costSummator.dataMat);

            CUDA_CHECKED_NO_ERROR();
        }
    }

    void computeMVBACost(CUDAEdgeVector &edges, Summator& costSummator) {
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].count);

            computeMVCostForEdge<<<grid, block, 0, stream>>>(edges[index], *costSummator.dataMat);

            CUDA_CHECKED_NO_ERROR();
        }
    }

}
