#include "hip/hip_runtime.h"
//
// Created by liulei on 2020/6/12.
//

#include "bundle_adjustment.cuh"

namespace rtf {
    constexpr CudaScalar kHuberWeight = 1.2;
    __device__ CudaScalar computeHuberWeight(CudaScalar residual_x, CudaScalar residual_y, CudaScalar huber_parameter) {
        CudaScalar squared_residual = residual_x * residual_x + residual_y * residual_y;
        return (squared_residual < huber_parameter * huber_parameter) ? 1 : (huber_parameter / sqrtf(squared_residual));
    }

    __device__ CudaScalar ComputeHuberCost(CudaScalar residual_x, CudaScalar residual_y, CudaScalar huber_parameter) {
        CudaScalar squared_residual = residual_x * residual_x + residual_y * residual_y;
        if (squared_residual < huber_parameter * huber_parameter) {
            return 0.5 * squared_residual;
        } else {
            return huber_parameter * (sqrtf(squared_residual) - 0.5 * huber_parameter);
        }
    }

    __device__ void projectJacobi(float3x3 k, CudaScalar* point, CudaScalar* proJacobi) {
        CudaScalar x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        CudaScalar y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        CudaScalar z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        proJacobi[0] = z*k(0,0)-x*k(2,0)/(z*z);
        proJacobi[1] = z*k(0,1)-x*k(2,1)/(z*z);
        proJacobi[2] = z*k(0,2)-x*k(2,2)/(z*z);
        proJacobi[3] = z*k(1,0)-y*k(2,0)/(z*z);
        proJacobi[4] = z*k(1,1)-y*k(2,1)/(z*z);
        proJacobi[5] = z*k(1,2)-y*k(2,2)/(z*z);
    }

    __device__ void transformPointOnlyRotation(float4x4 T, CudaScalar* point, CudaScalar *tPoint) {
        CudaScalar x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2];
        CudaScalar y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2];
        CudaScalar z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2];

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }


    __device__ void transformPoint(float4x4 T, CudaScalar* point, CudaScalar *tPoint) {
        CudaScalar x = T(0,0)*point[0] + T(0,1)*point[1] + T(0,2)*point[2] + T(0,3);
        CudaScalar y = T(1,0)*point[0] + T(1,1)*point[1] + T(1,2)*point[2] + T(1,3);
        CudaScalar z = T(2,0)*point[0] + T(2,1)*point[1] + T(2,2)*point[2] + T(2,3);

        tPoint[0] = x;
        tPoint[1] = y;
        tPoint[2] = z;
    }

    __device__ void projectPoint(float3x3 k, CudaScalar* point, CudaScalar* pixel) {
        CudaScalar x = k(0,0)*point[0]+k(0,1)*point[1]+k(0,2)*point[2];
        CudaScalar y = k(1,0)*point[0]+k(1,1)*point[1]+k(1,2)*point[2];
        CudaScalar z = k(2,0)*point[0]+k(2,1)*point[1]+k(2,2)*point[2];

        pixel[0] = x/z;
        pixel[1] = y/z;
    }

    __device__ void hatMatrix(CudaScalar* point, CudaScalar* hat) {
        hat[0] = 0;
        hat[1] = -point[2];
        hat[2] = point[1];
        hat[3] = point[2];
        hat[4] = 0;
        hat[5] = -point[0];
        hat[6] = -point[1];
        hat[7] = point[0];
        hat[8] = 0;
    }

    // x = u/fx*d-cx/fx*d, y=v/fy*d-cy/fy*d, z=d
    __device__ void unproject(float3x3 K, CudaScalar* pixel, CudaScalar *dst) {
        CudaScalar fx=K(0, 0), fy=K(1, 1), cx=K(0, 2), cy=K(1, 2);
        dst[0] = pixel[2]*(pixel[0]-cx)/fx;
        dst[1] = pixel[2]*(pixel[1]-cy)/fy;
        dst[2] = pixel[2];
    }

    __device__ void computeJacobi(CudaScalar* proJacobi, CudaScalar* hat, CudaScalar* jacobi) {
        // for R
        jacobi[0] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[1] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[2] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[6] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[7] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[8] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);

        // for t
        jacobi[3] = proJacobi[0];
        jacobi[4] = proJacobi[1];
        jacobi[5] = proJacobi[2];

        jacobi[9] = proJacobi[3];
        jacobi[10] = proJacobi[4];
        jacobi[11] = proJacobi[5];
    }


    __device__ void computeDeltaLie(CudaScalar *jacobi, CudaScalar *residual, CudaScalar* delta) {
        delta[0] = fabs(residual[0]/jacobi[0]+residual[1]/jacobi[6]);
        delta[1] = fabs(residual[0]/jacobi[1]+residual[1]/jacobi[7]);
        delta[2] = fabs(residual[0]/jacobi[2]+residual[1]/jacobi[8]);
        delta[3] = fabs(residual[0]/jacobi[3]+residual[1]*jacobi[9]); // jacobi is 0
        delta[4] = fabs(residual[0]*jacobi[4]+residual[1]/jacobi[10]);
        delta[5] = fabs(residual[0]/jacobi[5]+residual[1]/jacobi[11]);

    }



    __global__ void computeCostAndJacobi(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator, CUDAPtrs hSummator, CUDAPtrs mSummator, CUDAPtrs bSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;
        if(mask[index]) {
            CudaScalar point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)}
            , rePixel[2], transPoint[3], rotatePoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
            // copy point and pixel
            transformPoint(T, point, transPoint);

            // compute jacobi
            projectJacobi(K, transPoint, proJacobi);
            transformPointOnlyRotation(T, point, rotatePoint);
            hatMatrix(rotatePoint, hatMat);
            computeJacobi(proJacobi, hatMat, jacobi);

            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            CudaScalar weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
            CudaScalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
            // compute H,M,b
            CudaScalar * H = hSummator.data+index*36;
            CudaScalar * M = mSummator.data+index*6;
            CudaScalar * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
                }
                M[i] = weight*H[i*6+i];
                b[i] = -weight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
            }
        }else {
            // compute H,M,b
            costSummator.data[index]=0;
            CudaScalar * H = hSummator.data+index*36;
            CudaScalar * M = mSummator.data+index*6;
            CudaScalar * b = bSummator.data+index*6;
            for(int i=0; i<6; i++) {
                for(int j=0; j<6; j++) {
                    H[j*6+i] = 0;
                }
                M[i] = 0;
                b[i] = 0;
            }
        }

    }

    __global__ void computeCost(CUDAPtrs points, CUDAPtrs pixels, float4x4 T, float3x3 K, CUDAPtrc mask, CUDAPtrs costSummator) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=points.getRows()) return;

        if(mask[index]) {
            CudaScalar point[3]={points(index, 0), points(index, 1), points(index, 2)},
                    pixel[2] = {pixels(index, 0), pixels(index, 1)};

            CudaScalar rePixel[2], transPoint[3], residual[2];
            transformPoint(T, point, transPoint);
            projectPoint(K, transPoint, rePixel);
            // compute residual and cost
            residual[0] = rePixel[0] - pixel[0];
            residual[1] = rePixel[1] - pixel[1];

            CudaScalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

            costSummator.data[index]=cost;
        }else {
            costSummator.data[index] = 0;
        }
    }


    void computeBACostAndJacobi(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator, Summator& hSummator, Summator& mSummator, Summator& bSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCostAndJacobi<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat, *hSummator.dataMat, *mSummator.dataMat, *bSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    void computeBACost(CUDAMatrixs& objectPoints, CUDAMatrixs& tarPixels, float4x4& T, float3x3& K, CUDAMatrixc& mask, Summator& costSummator) {
        long n = objectPoints.getRows();
        // invoke kernel
        CUDA_LINE_BLOCK(n);

        computeCost<<<grid, block, 0, stream>>>(objectPoints, tarPixels, T, K, mask, *costSummator.dataMat);

        CUDA_CHECKED_NO_ERROR();
    }

    __global__ void computerInliersKernel(CUDAPtrs cost, CUDAPtrc inliers, CudaScalar th) {
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        if(index>=cost.getRows()) return;
        inliers.setIndex(index, cost[index]>0&&cost[index]<th);
    }

    void computerInliers(Summator& costSummator, CUDAMatrixc& inliers, CudaScalar th) {
        // invoke kernel
        CUDA_LINE_BLOCK(costSummator.length);

        computerInliersKernel<<<grid, block, 0, stream>>>(*costSummator.dataMat, inliers, th);

        CUDA_CHECKED_NO_ERROR();
    }

    __device__ void composeJacobi(CudaScalar* proJacobi, CudaScalar* hat, CudaScalar* jacobi) {
        // for t
        jacobi[0] = proJacobi[0];
        jacobi[1] = proJacobi[1];
        jacobi[2] = proJacobi[2];

        jacobi[6] = proJacobi[3];
        jacobi[7] = proJacobi[4];
        jacobi[8] = proJacobi[5];

        // for R
        jacobi[3] = -(proJacobi[0]*hat[0]+proJacobi[1]*hat[3]+proJacobi[2]*hat[6]);
        jacobi[4] = -(proJacobi[0]*hat[1]+proJacobi[1]*hat[4]+proJacobi[2]*hat[7]);
        jacobi[5] = -(proJacobi[0]*hat[2]+proJacobi[1]*hat[5]+proJacobi[2]*hat[8]);

        jacobi[9] = -(proJacobi[3]*hat[0]+proJacobi[4]*hat[3]+proJacobi[5]*hat[6]);
        jacobi[10] = -(proJacobi[3]*hat[1]+proJacobi[4]*hat[4]+proJacobi[5]*hat[7]);
        jacobi[11] = -(proJacobi[3]*hat[2]+proJacobi[4]*hat[5]+proJacobi[5]*hat[8]);
    }

    __device__ void computeHMb(CUDALMSummators summators, long index, CudaScalar weight, CudaScalar* jacobi, CudaScalar* residual, CudaScalar jacobiWeight) {
        CudaScalar * H = summators.H.data+index*36;
        CudaScalar * M = summators.M.data+index*6;
        CudaScalar * b = summators.b.data+index*6;
        for(int i=0; i<6; i++) {
            for(int j=0; j<6; j++) {
                H[j*6+i] = jacobi[i]*jacobi[j] + jacobi[i+6]*jacobi[j+6];
            }
            M[i] = weight*H[i*6+i];
            b[i] = -weight*jacobiWeight*(jacobi[i]*residual[0]+jacobi[i+6]*residual[1]);
        }
    }


    __global__ void computeMVCostAndJacobiForEdge(CUDAEdge edge, CUDALMSummators summatorsX, CUDALMSummators summatorsY, CUDALMSummators deltaSummators, CUDAPtrs costSummator) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        long sumIndexX = edge.sumIndexX+index;
        long sumIndexY = edge.sumIndexY+index;
        long costIndex = edge.costIndex+index;
        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        float3x3 intrinsicX = edge.intrinsicX;
        float3x3 intrinsicY = edge.intrinsicY;
        float4x4 transform = edge.transform;

        if(index>=kx.getRows()) return;

        CudaScalar point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};
        CudaScalar rePixel[2], transPoint[3], proJacobi[6], hatMat[9], residual[2], jacobi[12];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        // compute jacobi
        projectJacobi(intrinsicX, transPoint, proJacobi);
        hatMatrix(transPoint, hatMat);
        composeJacobi(proJacobi, hatMat, jacobi);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        CudaScalar weight = computeHuberWeight(residual[0], residual[1], kHuberWeight);
        CudaScalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

        costSummator.data[costIndex]=cost;
        // compute H,M,b
        computeHMb(summatorsX, sumIndexX, weight, jacobi, residual, 1.0);
        computeHMb(summatorsY, sumIndexY, weight, jacobi, residual, -1.0);
        computeHMb(deltaSummators, index, weight, jacobi, residual, 1.0);
    }

    __global__ void computeMVCostForEdge(CUDAEdge edge, CUDAPtrs costSummator) {
        // obtain parameters from
        long index = threadIdx.x + blockIdx.x*blockDim.x;

        long costIndex = edge.costIndex+index;
        CUDAPtrs kx = edge.kx;
        CUDAPtrs ky = edge.ky;
        float3x3 intrinsicX = edge.intrinsicX;
        float3x3 intrinsicY = edge.intrinsicY;
        float4x4 transform = edge.transform;

        if(index>=kx.getRows()) return;

        CudaScalar point[3]={ky(index, 0), ky(index, 1), ky(index, 2)},
                pixel[2] = {kx(index, 0), kx(index, 1)};

        CudaScalar rePixel[2], transPoint[3], residual[2];
        unproject(intrinsicY, point, transPoint);
        transformPoint(transform, transPoint, transPoint);

        projectPoint(intrinsicX, transPoint, rePixel);
        // compute residual and cost
        residual[0] = rePixel[0] - pixel[0];
        residual[1] = rePixel[1] - pixel[1];

        CudaScalar cost = ComputeHuberCost(residual[0], residual[1], kHuberWeight);

        costSummator.data[costIndex]=cost;
    }


    void computeMVBACostAndJacobi(CUDAEdgeVector &edges, CUDAVector<CUDALMSummators>& gtSummators, CUDAVector<CUDALMSummators>& deltaSummators, Summator& costSummator) {
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].count);

            computeMVCostAndJacobiForEdge<<<grid, block, 0, stream>>>(edges[index], gtSummators[edges[index].indexX], gtSummators[edges[index].indexY], deltaSummators[index], *costSummator.dataMat);

            CUDA_CHECKED_NO_ERROR();
        }
    }

    void computeMVBACost(CUDAEdgeVector &edges, Summator& costSummator) {
        for(long index=0; index<edges.getNum(); index++) {
            CUDA_LINE_BLOCK(edges[index].count);

            computeMVCostForEdge<<<grid, block, 0, stream>>>(edges[index], *costSummator.dataMat);

            CUDA_CHECKED_NO_ERROR();
        }
    }

}
