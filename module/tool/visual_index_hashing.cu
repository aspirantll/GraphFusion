#include "hip/hip_runtime.h"
//
// Created by liulei on 2020/10/15.
//
#include "visual_index_hashing.cuh"

namespace rtf {


    __global__ void wordsCountKernel(CUDAPtrArray<CUDABoW> voc, CUDAPtru cur, CUDAPtru counts) {
        const int tid = threadIdx.x;
        const int index = threadIdx.x + blockIdx.x*blockDim.x;
        if(index>=voc.getNum()) return;

        CUDABoW bow = voc[index];
        const unsigned int imageId = bow.imageId;
        CUDAPtru ref = bow.words;

        const int curLength = cur.size();
        const int refLength = ref.size();
        // copy cur to shared memory
        extern __shared__ unsigned int curWords[];
        for(int i=tid; i<curLength; i+=blockDim.x) {
            curWords[i] = cur[i];
        }
        __syncthreads();
        unsigned int count = 0;
        int i=0, j=0;
        while(i<curLength&&j<refLength) {
            unsigned int curWordId = curWords[i];
            unsigned int refWordId = ref[j];
            if(curWordId==refWordId) {
                i++;
                j++;
                count++;
            }else if(curWordId>refWordId) {
                j++;
            }else {
                i++;
            }
        }

        counts.setIndex(index, count);
    }


    void wordsCount(CUDAPtrArray<CUDABoW>& voc, CUDAArrayu& cur, CUDAArrayu& wordCounts) {
        int length = voc.getNum();
        CUDA_LINE_BLOCK(length);
        wordsCountKernel<<<grid, block, sizeof(unsigned int)*cur.getSizes(), stream>>>(voc, cur, wordCounts);
        CUDA_CHECKED_NO_ERROR();
    }
}
